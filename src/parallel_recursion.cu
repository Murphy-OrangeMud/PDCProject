#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>
#include <iostream>
#include <fstream>
#include <ctime>
#include <cstdio>
using namespace std;

# define N 1024

__global__ void 
Forward(double *u, double *l, double *d, double *rhs, int *p, int cellSize, int *s, int low, int high) {
    __synchronize();
    int b = threadIdx.x;
    for (int i = b + low; i <= high; i += N) {
        if (s[i]) continue;
        rhs[i] -= l[i] * rhs[p[i]];
        rhs[i] /= d[i];
    }
    __synchronize();
}

__global__ void 
Backward(double *u, double *l, double *d, double *rhs, int *p, int cellSize, int *s, int low, int high, int &nlow, int &nhigh) {
    int b = threadIdx.x;
    int *ns;
    hipMalloc(&ns, cellSize * sizeof(int));
    hipMemcpy(ns, s, hipMemcpyDefault);
    __synchronize();
    for (int i = high - b; i >= low; i -= N) {
        if (s[i]) continue;
        factor = u[i] / d[i];
        d[p[i]] -= factor * l[i];
        rhs[p[i]] -= factor * rhs[i];
        ns[p[i]]--;
        ns[i]--;
        nhigh = max(nhigh, p[i]);
        nlow = min(nlow, p[i]);
    }
    __synchronize();
    return *ns;
}

void Hines(double *u, double *l, double *d, double *rhs, int *p, int cellSize, int *s, int low, int high) {
    if (high == 0) {
        rhs[0] /= d[0];
        return;
    }
    int nlow = cellSize - 1, nhigh = 0;
    int *ns = Backward<<<1, N>>>(u, l, d, rhs, p, cellSize, s, low, high, nlow, nhigh);
    Hines(u, l, d, rhs, p, cellSize, ns, nlow, nhigh);
    Forward<<<1, N>>>(u, l, d, rhs, p, cellSize, s, low, high);
}

int cellSize;
int main(int argc, char *argv[]) {
    int devID = 0;
    hipSetDevice(devID);

    hipError_t error;
    cudaDevice Prop deviceProp;
    error = hipGetDevice(&devID);

    if (error != hipSuccess) {
        cout << "hipGetDevice returned error " << hipGetErrorString(error) << " (code " << error << "), line(" << __LINE__ << endl;
        exit(0); 
    }

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.computeMode == hipComputeModeProhibited) {
        cerr << "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n";
        exit(EXIT_SUCCESS);
    }

    if (error != hipSuccess) {
        cout << "hipGetDeviceProperties returned error " << hipGetErrorString(error) << " (code " << error << "), line(" << __LINE__ << endl;
    } else {
        cout << "GPU Device " << devID << ": " << deviceProp.name << " with compute capability " << deviceProp.major << "." << deviceProp.minor << endl << endl;
    }

    char *path = argv[1];
    fstream file(path, ios::in);
    int dsize = (cellSize + 5) * sizeof(double);
    int isize = (cellSize + 5) * sizeof(int);
    file >> cellSize;
    double *u, *l, *d, *rhs;
    int *p, *s;
    hipMalloc(&u, dsize);
    hipMalloc(&l, dsize);
    hipMalloc(&d, dsize);
    hipMalloc(&rhs, dsize);
    hipMalloc(&p, isize);
    hipMalloc(&s, isize);
    hipMemset(s, 0, isize);
    int idx;
    int low = cellSize - 1, high = 0;
    for (int i = 0; i < cellSize; i++) {
        file >> idx >> u[i] >> l[i] >> rhs[i] >> d[i] >> p[i];
        if (!i) continue;
        s[p[i]]++;
    }
    for (int i = 0; i < cellSize; i++) {
        if (s[i] == 0) {
            low = min(low, i);
            high = max(high, i);
        }
    }
    unsigned start = clock();
    Hines(u, l, d, rhs, p, cellSize, s, low, high);
    unsigned end = clock();
    cout << "time cost: " << (float)(end-start)*1000.0/CLOCKS_PER_SEC << "ms" << endl;
    fstream outfile(argv[2], ios::out);
    for (int i = 0; i < cellSize; i++) {
        outfile << i << " " << u[i] << " " << l[i] << " " << rhs[i] << " " << d[i] << endl;
    }
    /*
    delete [] u;
    cout << "u deleted\n";
    delete [] l;
    cout << "l deleted\n";
    delete [] d;
    cout << "d deleted\n";
    delete [] rhs;
    cout << "rhs deleted\n";
    delete [] p;
    cout << "p deleted\n";
    */
    return 0;
}